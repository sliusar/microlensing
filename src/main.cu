#include "hip/hip_runtime.h"
#include <common.cuh>
#include <config.cuh>
#include <kernels.cuh>
#include "timer.h"

using namespace std;

#define hipDeviceScheduleBlockingSync 0x04
#define hipEventBlockingSync 0x01

#define CUDA_BLOCK_SIZE 1024
#define CUDA_BLOCK_SIZE_2d 32

int verbose = 0;

int write_image(char* filename, int* image, Configuration c) {
  FILE *fp = fopen(filename, "wb");
  if(fp == NULL) {
    cout << "Error opening the file " << filename << endl;
    return -1;
  }
  fwrite((const void*)&c.image_width, sizeof(c.image_width), 1, fp);
  fwrite((const void*)&c.image_height, sizeof(c.image_height), 1, fp);
  fwrite((const void*)&c.image_y1_left, sizeof(c.image_y1_left), 1, fp);
  fwrite((const void*)&c.image_y1_right, sizeof(c.image_y1_right), 1, fp);
  fwrite((const void*)&c.image_y2_bottom, sizeof(c.image_y2_bottom), 1, fp);
  fwrite((const void*)&c.image_y2_top, sizeof(c.image_y2_top), 1, fp);
  fwrite((const void*)&image[0], sizeof(image[0]), c.image_width * c.image_height, fp);
  fclose(fp);
  return 0;
}

int write_lc(char* filename, float* lc, Configuration c) {
  FILE *fp = fopen(filename, "wb");
  if(fp == NULL) {
    cout << "Error opening the file " << filename << endl;
    return -1;
  }
  int col_count = 3 + (c.nLCcolumns - 2)/2;
  fwrite((const void*)&col_count, sizeof(col_count), 1, fp);
  fwrite((const void*)&c.nLCsteps, sizeof(c.nLCsteps), 1, fp);
  fwrite((const void*)&c.source_size, sizeof(c.source_size), 1, fp);
  fwrite((const void*)&c.eccentricity, sizeof(c.eccentricity), 1, fp);
  int counter = 0;
  for (float t = 0.0; t < c.lc_dist_max; t = t + c.lc_dist_step) {
    fwrite((const void*)&t, sizeof(float), 1, fp);
    fwrite((const void*)&lc[counter + 0 * c.nLCsteps], sizeof(float), 1, fp);
    fwrite((const void*)&lc[counter + 1 * c.nLCsteps], sizeof(float), 1, fp);
    for (int i = 2; i < c.nLCcolumns; i+=2) {
      float k = lc[counter + (i + 1) * c.nLCsteps] / lc[counter + i * c.nLCsteps];
      fwrite((const void*)&k, sizeof(float), 1, fp);
    }
  counter++;
}


  
  fclose(fp);
  return 0;
}

double getCurrentTimestamp() {
  struct timeval time_now{};
  gettimeofday(&time_now, nullptr);
  return time_now.tv_sec + ((double)time_now.tv_usec / 1e6);
}

int estimateRaysCount(float R_rays, float dx_rays) {
  int counter = 0;
  for (float x1 = - R_rays; x1 <= R_rays; x1 += dx_rays) {
    for (float x2 = - R_rays; x2 <= R_rays; x2 += dx_rays) {
      if (distance(x1, x2) <= R_rays) counter++;
    }
  }
  return counter;
}

int main(const int argc, const char** argv) {
  double t0 = getCurrentTimestamp();
  if (argc != 2) {
    cerr << "Usage:\n\t" << argv[0] << " configuration.yaml" << endl;
    return 1;
  }

  char filename[64], output_folder[64];

  Configuration conf(argv[1]);
  conf.prepare_sources();
  conf.display();
  verbose = conf.verbose;
  if (conf.randomise_seed_number != 0) {
    long _seed = time(NULL);
    if (conf.randomise_seed_number > 0) _seed = conf.randomise_seed_number;
    cout << "Using " << _seed << " to seed the random generator" << endl;
    srand(_seed);
  }

  int _c = estimateRaysCount(conf.R_rays, conf.dx_rays);
  cout << "Print estimated rays count " << _c << " (previous " << conf.nRays << "). Adjusting value." << endl;
  conf.nRays = _c;

  int uls_bytes = conf.nMicrolenses * sizeof(Microlens);
  int rays_bytes = conf.nRays * sizeof(Ray);
  int image_bytes = conf.image_height * conf.image_width * sizeof(int);
  int lc_bytes = conf.nLCcolumns * conf.nLCsteps * sizeof(float);

  Microlens *microlenses = (Microlens*)malloc(uls_bytes);
  Ray *rays = (Ray*)malloc(rays_bytes);
  int *image = (int*)malloc(image_bytes);
  float *lc = (float*)malloc(lc_bytes);

  Microlens *ul_buf;
  Ray *rays_buf;
  int *image_buf;
  float *lc_buf;

  struct stat info;

  sprintf(output_folder, "./output/%s", conf.configuration_id.c_str());
  if( stat(output_folder , &info ) != 0 ) {
    if (mkdir(output_folder, 0755) != 0 && errno != EEXIST) {
      cerr << "Failed to create output folder " << output_folder << endl;
      return -1;
    }
  }

  hipMalloc(&image_buf, image_bytes);
  
  cout << "Creating rays field ... " << flush;
  StartTimer();
  hipMalloc(&rays_buf, rays_bytes);
  populateRays(rays, conf.nRays, conf.R_rays, conf.dx_rays);
  hipMemcpy(rays_buf, rays, rays_bytes, hipMemcpyHostToDevice);
  cout << GetElapsedTime() << "s" << endl;

  if (conf.lc_enabled) {
    cout << "Creating light curve placeholder ... " << flush;
    StartTimer();
    createLC(lc, conf);
    hipMalloc(&lc_buf, lc_bytes);
    hipMemcpy(lc_buf, lc, lc_bytes, hipMemcpyHostToDevice);
    cout << GetElapsedTime() << "s" << endl;
  }

  int nBlocksRays = ceil((float)conf.nRays / (float)CUDA_BLOCK_SIZE);
  int nBlocksImageW = ceil((float)conf.image_width / (float)CUDA_BLOCK_SIZE_2d);
  int nBlocksImageH = ceil((float)conf.image_height / (float)CUDA_BLOCK_SIZE_2d);

  cout << endl << "GPU Execution settings:" << endl;
  cout << "    nBlocksRays: " << nBlocksRays << endl;
  cout << "    nBlocksImage: " << nBlocksImageW << " x " << nBlocksImageH << endl;

  //From https://stackoverflow.com/questions/11888772/when-to-call-cudadevicesynchronize
  //  kernel1<<<X,Y>>>(...); // kernel start execution, CPU continues to next statement
  //  kernel2<<<X,Y>>>(...); // kernel is placed in queue and will start after kernel1 finishes, CPU continues to next statement
  //  hipMemcpy(...); // CPU blocks until memory is copied, memory copy starts only after kernel2 finishes

  ofstream outf;
  int counter = 0;
  float _t = 0;
  float t_raytracing = 0;
  float t_output = 0;
  float t_lc = 0;
  bool microlenses_set = false;
  
  for (float t = 0; t <= conf.t_max; t = t + conf.dt) {
    cout << endl << ">>> Iteration #" << ++counter << ", t=" << t << " (elapsed: " << getCurrentTimestamp() - t0 << "s)" << endl;

    memset(image, 0, image_bytes);
    hipMemcpy(image_buf, image, image_bytes, hipMemcpyHostToDevice);

    resetLC(lc, conf);
    hipMemcpy(lc_buf, lc, lc_bytes, hipMemcpyHostToDevice);
    
    if (conf.operation_mode == 1 || (conf.operation_mode == 0 && microlenses_set == false)) {
      microlenses_set = true;
      cout << "    Creating microlensing field ... " << flush;
      StartTimer();
      randomiseMicrolenses(microlenses, conf);  
      hipMalloc(&ul_buf, uls_bytes);
      hipMemcpy(ul_buf, microlenses, uls_bytes, hipMemcpyHostToDevice);
      cout << GetElapsedTime() << "s" << endl;
    }

    cout << "    [CUDA] Running ray tracing ... " << flush;
    StartTimer();
    deflectRays<<<nBlocksRays, CUDA_BLOCK_SIZE>>>(ul_buf, rays_buf, conf, t, image_buf, lc_buf); // compute ray deflections
    if (conf.save_rays) hipMemcpy(rays, rays_buf, rays_bytes, hipMemcpyDeviceToHost);
    if (conf.lc_enabled) hipMemcpy(lc, lc_buf, lc_bytes, hipMemcpyDeviceToHost);
    if (conf.save_images) hipMemcpy(image, image_buf, image_bytes, hipMemcpyDeviceToHost);
    hipError_t err = hipDeviceSynchronize();
    if(err != hipSuccess) {
      cerr << "Error running the deflectRays() kernel" << endl;
      return -1;
    }
    _t = GetElapsedTime();
    t_raytracing += _t;
    cout << _t << "s" << endl;
    
    if (conf.lc_enabled) {
      cout << "    [CUDA] Calculating light curves ... " << flush;
      StartTimer();
      calculateLCs<<<dim3(nBlocksImageW, nBlocksImageH), dim3(CUDA_BLOCK_SIZE_2d, CUDA_BLOCK_SIZE_2d)>>>(conf, image_buf, lc_buf); // compute lc
      hipMemcpy(lc, lc_buf, lc_bytes, hipMemcpyDeviceToHost);
      hipError_t err = hipDeviceSynchronize();
      if(err != hipSuccess) {
        cerr << "Error running the calculateLCs() kernel" << endl;
        return -1;
      }
      _t = GetElapsedTime();
      t_lc += _t;
      cout << _t << "s" << endl;
    }

    if (conf.save_rays) {
      sprintf(filename, "%s/rays_%.2f.dat", output_folder, t);
      cout << "    Writing rays data to " << filename << " ... " << flush;
      outf.open(filename);
      for (int i = 0; i <= conf.nRays; i++) {
        if (rays[i].x1 >= conf.image_y1_left && rays[i].x1 <= conf.image_y1_right && rays[i].x2 >= conf.image_y2_bottom && rays[i].x2 <= conf.image_y2_top) {
          outf << rays[i].x1 << " " << rays[i].x2 << endl;
        }
      }
      outf.close();
      cout << GetElapsedTime() << "s" << endl;
    }

    sprintf(filename, "%s/image_%.2f.dat", output_folder, t);
    if (conf.save_images) {
      StartTimer();
      cout << "    Writing image data to " << filename << " ... " << flush;
      write_image(filename, image, conf);
      _t = GetElapsedTime();
      t_output += _t;
      cout << _t << "s" << endl;  
    } else {
      cout << "    Skipping image data writing to " << filename << " ... " << endl;
    }

    
    if (conf.lc_enabled) {
      sprintf(filename, "%s/lc_%.2f.dat", output_folder, t);
      StartTimer();
      cout << "    Writing light curves data to " << filename << " ... " << flush;
      write_lc(filename, lc, conf);

#if DEBUG == true
      sprintf(filename, "%s/lc_%.2f.txt", output_folder, t);
      outf.open(filename);
      cout << "    Writing light curves data to " << filename << " ... " << flush;

      outf << "# - - - ";
      for (int i = 2; i < conf.nLCcolumns; i+= 2) {
        outf << lc[(conf.nLCsteps - 1) + (i + 1) * conf.nLCsteps] << "|" <<  lc[(conf.nLCsteps - 1) + i * conf.nLCsteps] << " ";
      }
      outf << endl;

      int counter = 0;
      outf << "# t y1 y2 ad(1) gauss(2) ld(3) pl(4) el(5) el_orth(6)" << endl;
      for (float t = 0.0; t < conf.lc_dist_max; t = t + conf.lc_dist_step) {
          outf << t << " " << lc[counter + 0 * conf.nLCsteps] << " " << lc[counter + 1 * conf.nLCsteps] << " ";
          for (int i = 2; i < conf.nLCcolumns; i+=2) {
            outf << lc[counter + (i + 1) * conf.nLCsteps] / lc[counter + i * conf.nLCsteps] << " ";
          }
          outf << endl;
        counter++;
      }

      outf.close();
#endif
      _t = GetElapsedTime();
      t_output += _t;
      cout << _t << "s" << endl;        
    }
  }

  free(microlenses);
  free(rays);
  free(image);
  free(lc);
  
  hipFree(ul_buf);
  hipFree(lc_buf);
  hipFree(rays_buf);
  hipFree(image_buf);

  cout << endl << ">>> Run summary:" << endl;
  cout << " Raytracing time: " << t_raytracing << "s (mean: " << t_raytracing/counter << "s)" << endl;
  cout << " Light curve calculation time: " << t_lc << "s (mean: " << t_lc/counter << "s)" << endl;
  cout << " Output time: " << t_output << "s (mean: " << t_output / counter << "s)" << endl;
  cout << " Total time: " << getCurrentTimestamp() - t0 << "s" << endl;

  return 0;
}
