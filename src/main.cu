#include "hip/hip_runtime.h"
#include <common.cuh>
#include <config.cuh>
#include <kernels.cuh>
#include "timer.h"

using namespace std;

#define hipDeviceScheduleBlockingSync 0x04
#define hipEventBlockingSync 0x01

#define CUDA_BLOCK_SIZE 1024
#define CUDA_BLOCK_SIZE_2d 32

#define LC_COLUMNS 10

int write_image(char* filename, int* image, int image_size) {
  ofstream wf(filename, ios::out | ios::binary);
  if(!wf) {
     cout << "Cannot open file " << filename << endl;
     return 1;
  }
  wf.write((char *)&image_size, sizeof(image_size));
  wf.close();
  if(!wf.good()) {
     cout << "Error occurred at writing of" << filename << endl;
     return 1;
  }
}

double getCurrentTimestamp() {
  struct timeval time_now{};
  gettimeofday(&time_now, nullptr);
  return time_now.tv_sec + ((double)time_now.tv_usec / 1e6);
}

int estimateRaysCount(float R_rays, float dx_rays) {
  int counter = 0;
  for (float x1 = - R_rays; x1 <= R_rays; x1 += dx_rays) {
    for (float x2 = - R_rays; x2 <= R_rays; x2 += dx_rays) {
      if (distance(x1, x2) <= R_rays) counter++;
    }
  }
  return counter;
}

int main(const int argc, const char** argv) {
  double t0 = getCurrentTimestamp();
  if (argc != 2) {
    cerr << "Usage:\n\t" << argv[0] << " configuration.yaml" << endl;
    return 1;
  }

  char filename[64], output_folder[64];

  Configuration conf(argv[1]);
  conf.prepare_sources();
  conf.display();
  if (conf.randomise_seed_number != 0) {
    long _seed = time(NULL);
    if (conf.randomise_seed_number > 0) _seed = conf.randomise_seed_number;
    cout << "Using " << _seed << " to seed the random generator" << endl;
    srand(_seed);
  }

  int _c = estimateRaysCount(conf.R_rays, conf.dx_rays);
  cout << "Print estimated rays count " << _c << " (previous " << conf.nRays << "). Adjusting value." << endl;
  conf.nRays = _c;

  int uls_bytes = conf.nMicrolenses * sizeof(Microlens);
  int rays_bytes = conf.nRays * sizeof(Ray);
  int image_bytes = conf.image_height * conf.image_width * sizeof(int);
  int lc_bytes = LC_COLUMNS * conf.nLCsteps * sizeof(float);

  Microlens *microlenses = (Microlens*)malloc(uls_bytes);
  Ray *rays = (Ray*)malloc(rays_bytes);
  int *image = (int*)malloc(image_bytes);
  float *lc = (float*)malloc(lc_bytes);

  Microlens *ul_buf;
  Ray *rays_buf;
  int *image_buf;
  float *lc_buf;

  struct stat info;

  sprintf(output_folder, "./output/%s", conf.configuration_id.c_str());
  if( stat(output_folder , &info ) != 0 ) {
    if (mkdir(output_folder, 0755) != 0 && errno != EEXIST) {
      cerr << "Failed to create output folder " << output_folder << endl;
      return -1;
    }
  }

  hipMalloc(&image_buf, image_bytes);
  
  cout << "Creating microlensing field ... " << flush;
  StartTimer();
  randomiseMicrolenses(microlenses, conf);  
  hipMalloc(&ul_buf, uls_bytes);
  hipMemcpy(ul_buf, microlenses, uls_bytes, hipMemcpyHostToDevice);
  cout << GetElapsedTime() << "s" << endl;

  cout << "Creating rays field ... " << flush;
  StartTimer();
  hipMalloc(&rays_buf, rays_bytes);
  populateRays(rays, conf.nRays, conf.R_rays, conf.dx_rays);
  hipMemcpy(rays_buf, rays, rays_bytes, hipMemcpyHostToDevice);
  cout << GetElapsedTime() << "s" << endl;

  if (conf.lc_enabled) {
    cout << "Creating light curve placeholder ... " << flush;
    StartTimer();
    createTrajectory(lc, conf);
    hipMalloc(&lc_buf, lc_bytes);
    hipMemcpy(lc_buf, lc, lc_bytes, hipMemcpyHostToDevice);
    cout << GetElapsedTime() << "s" << endl;
  }

  int nBlocksRays = ceil((float)conf.nRays / (float)CUDA_BLOCK_SIZE);
  int nBlocksImageW = ceil((float)conf.image_width / (float)CUDA_BLOCK_SIZE_2d);
  int nBlocksImageH = ceil((float)conf.image_height / (float)CUDA_BLOCK_SIZE_2d);

  cout << endl << "GPU Execution settings:" << endl;
  cout << "    nBlocksRays: " << nBlocksRays << endl;
  cout << "    nBlocksImage: " << nBlocksImageW << " x " << nBlocksImageH << endl;

  //From https://stackoverflow.com/questions/11888772/when-to-call-cudadevicesynchronize
  //  kernel1<<<X,Y>>>(...); // kernel start execution, CPU continues to next statement
  //  kernel2<<<X,Y>>>(...); // kernel is placed in queue and will start after kernel1 finishes, CPU continues to next statement
  //  hipMemcpy(...); // CPU blocks until memory is copied, memory copy starts only after kernel2 finishes

  ofstream outf;
  int counter = 0;
  float _t = 0;
  float t_raytracing = 0;
  float t_output = 0;
  float t_lc = 0;
  
  for (float t = 0; t <= conf.t_max; t = t + conf.dt) {
    memset(image, 0, image_bytes);
    hipMemcpy(image_buf, image, image_bytes, hipMemcpyHostToDevice);

    resetTrajectory(lc, conf);
    hipMemcpy(lc_buf, lc, lc_bytes, hipMemcpyHostToDevice);

    cout << endl << ">>> Iteration #" << ++counter << ", t=" << t << " (elapsed: " << getCurrentTimestamp() - t0 << "s)" << endl;
    
    cout << "    [CUDA] Running ray tracing ... " << flush;
    StartTimer();
    deflectRays<<<nBlocksRays, CUDA_BLOCK_SIZE>>>(ul_buf, rays_buf, conf, t, image_buf, lc_buf); // compute ray deflections
    if (conf.output_rays) hipMemcpy(rays, rays_buf, rays_bytes, hipMemcpyDeviceToHost);
    if (conf.lc_enabled) hipMemcpy(lc, lc_buf, lc_bytes, hipMemcpyDeviceToHost);
    if (conf.save_images) hipMemcpy(image, image_buf, image_bytes, hipMemcpyDeviceToHost);
    hipError_t err = hipDeviceSynchronize();
    if(err != hipSuccess) {
      cerr << "Error running the deflectRays() kernel" << endl;
      return -1;
    }
    _t = GetElapsedTime();
    t_raytracing += _t;
    cout << _t << "s" << endl;
    
    if (conf.lc_enabled) {
      cout << "    [CUDA] Calculating light curves ... " << flush;
      StartTimer();
      calculateLCs<<<dim3(nBlocksImageW, nBlocksImageH), dim3(CUDA_BLOCK_SIZE_2d, CUDA_BLOCK_SIZE_2d)>>>(conf, image_buf, lc_buf); // compute lc
      hipMemcpy(lc, lc_buf, lc_bytes, hipMemcpyDeviceToHost);
      hipError_t err = hipDeviceSynchronize();
      if(err != hipSuccess) {
        cerr << "Error running the calculateLCs() kernel" << endl;
        return -1;
      }
      _t = GetElapsedTime();
      t_lc += _t;
      cout << _t << "s" << endl;
    }

    if (conf.output_rays) {
      sprintf(filename, "%s/rays_%.2f.dat", output_folder, t);
      cout << "    Writing data to " << filename << " ... " << flush;
      outf.open(filename);
      for (int i = 0; i <= conf.nRays; i++) {
        if (rays[i].x1 >= conf.image_y1_left && rays[i].x1 <= conf.image_y1_right && rays[i].x2 >= conf.image_y2_bottom && rays[i].x2 <= conf.image_y2_top) {
          outf << rays[i].x1 << " " << rays[i].x2 << endl;
        }
      }
      outf.close();
      cout << GetElapsedTime() << "s" << endl;
    }

    sprintf(filename, "%s/image_%.2f.dat", output_folder, t);
    if (conf.save_images) {
      StartTimer();
      cout << "    Writing data to " << filename << " ... " << flush;
      outf.open(filename);
      outf << "# image (" << conf.image_width << ", " << conf.image_height << ")" << endl;
      outf << "# x in (" << conf.image_y1_left << ", " << conf.image_y1_right << ")" << endl;
      outf << "# y in (" << conf.image_y2_bottom << ", " << conf.image_y2_top << ")" << endl;

      for (int j = 0; j < conf.image_height; j++) {
        for (int i = 0; i < conf.image_width; i++) {
          outf << image[i * conf.image_width + j] << endl;
        }
      }
      outf.close();
      _t = GetElapsedTime();
      t_output += _t;
      cout << _t << "s" << endl;  
    } else {
      cout << "    Skipping writing data to " << filename << " ... " << endl;
    }

    sprintf(filename, "%s/lc_%.2f.dat", output_folder, t);
    if (conf.lc_enabled) {
      StartTimer();
      cout << "    Writing light curves data to " << filename << " ... " << flush;
      outf.open(filename);
      int c = conf.nLCsteps;
      for (int i = 0; i < c; i++) {
        outf << lc[i + 0 * c] << " " << lc[i + 1 * c] << " ";
        outf << lc[i + 2 * c] << " " << lc[i + 3 * c] << " ";
        outf << lc[i + 4 * c] << " " << lc[i + 5 * c] << " ";
        outf << lc[i + 6 * c] << " " << lc[i + 7 * c] << " ";
        outf << lc[i + 8 * c] << " " << lc[i + 9 * c] << " ";
        outf << endl;
      }
      outf.close();
      _t = GetElapsedTime();
      t_output += _t;
      cout << _t << "s" << endl;        
    }
  }

  free(microlenses);
  free(rays);
  free(image);
  free(lc);
  
  hipFree(ul_buf);
  hipFree(lc_buf);
  hipFree(rays_buf);
  hipFree(image_buf);

  cout << endl << ">>> Run summary:" << endl;
  cout << "    Raytracing time: " << t_raytracing << "s (mean: " << t_raytracing/counter << "s)" << endl;
  cout << "    Output time: " << t_output << "s (mean: " << t_output / counter << "s)" << endl;
  cout << "    Total time: " << getCurrentTimestamp() - t0 << "s" << endl;

  return 0;
}
