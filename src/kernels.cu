#include "hip/hip_runtime.h"
#include <common.cuh>
#include <config.cuh>

float distance(float x, float y, float center_x, float center_y) {
  return sqrt(pow(x - center_x, 2) + pow(y - center_y, 2));
}

float distance(float x, float y) {
  return distance(x, y, 0, 0);
}

void randomiseMicrolenses(Microlens *ul, int n, float R) {
  
  for (int i = 0; i < n; i++) {
    float x1 = 2 * R * (rand() / (float)RAND_MAX) - R;
    float x2 = 2 * R * (rand() / (float)RAND_MAX) - R;

    while (distance(x1, x2) > R) {
      x1 = 2 * R * (rand() / (float)RAND_MAX) - R;
      x2 = 2 * R * (rand() / (float)RAND_MAX) - R;
    }
    ul[i] = {.x1 = x1, .x2 = x2, .v1 = 0.0, .v2 = 0.0, .m = 1.0 };
  }

  float speed_range_radius = 1;
  for (int i = 0; i < n; i++) {
    float v1 = speed_range_radius * (rand() / (float)RAND_MAX) - speed_range_radius;
    float v2 = speed_range_radius * (rand() / (float)RAND_MAX) - speed_range_radius;
    ul[i].v1 = v1;
    ul[i].v2 = v2;
  }
}

void populateRays(Ray *rays, int nRays, float R_rays, float dx_rays) {
    for (int i = 0; i < nRays; i++) rays[i] = { .x1 = 0, .x2 = 0 };
    //int counter = 0;
    //for (float x1 = - R_rays; x1 <= R_rays; x1 += dx_rays) {
    //  for (float x2 = - R_rays; x2 <= R_rays; x2 += dx_rays) {
    //    if (distance(x1, x2) <= R_rays && counter < nRays) rays[counter++] = {.x1 = x1, .x2 = x2 };
    //  }
    //}
}

void createTrajectory(float *lc_trajectory, const Configuration conf) {
    int counter = 0;
    float l = sqrt(pow(conf.lc_end_y1 - conf.lc_start_y1, 2) + pow(conf.lc_end_y2 - conf.lc_start_y2, 2));
    float step_y1 = conf.lc_step * (conf.lc_end_y1 - conf.lc_start_y1)/l;
    float step_y2 = conf.lc_step * (conf.lc_end_y2 - conf.lc_start_y2)/l;
    for (float y1 = conf.lc_start_y1, y2 = conf.lc_start_y2; (y1 < conf.lc_end_y1 && y2 < conf.lc_end_y2); y1+=step_y1, y2+=step_y2) {
        //i_y1 = round((y1 - conf.image_y1_left) / conf.image_pixel_y1_size);
        //i_y2 = round((y2 - conf.image_y2_bottom) / conf.image_pixel_y2_size);
        lc_trajectory[counter + 0 * conf.nLCsteps] = y1; // Y1 coordinate
        lc_trajectory[counter + 1 * conf.nLCsteps] = y2; // Y2 coordinate
        lc_trajectory[counter + 2 * conf.nLCsteps] = 0.0; // Gauss amplitude normalization value
        lc_trajectory[counter + 3 * conf.nLCsteps] = 0.0; // Gauss amplitude value
        counter++;
    }
}

void printLC(float *lc, int c) {
  cout << endl << "Y1\tY2\tNorm\tVal" << endl;
  for (int i = 0; i < c; i++) {
    cout << lc[i + 0 * c] << "\t" << lc[i + 1 * c] << "\t" << lc[i + 2 * c] << "\t" << lc[i + 3 * c] << endl;
  }
}

__device__ float dst2_inv(float x, float y) {
  return rhypotf(x, y) * rhypotf(x, y);
}

__device__ float dst(float x, float y) {
  return hypotf(x, y);
}

__device__ float dst(float x1, float y1, float x2, float y2) {
  return hypotf(x1 - x2, y1 - y2);
}

__device__ float H(float a, float x) {
  return (x >= a) ? 1.0 : 0.0;
}

__global__ void deflectRays(Microlens *uls, Ray *rays, const Configuration c, const float t, int *image, float* lc) {
  int ri = blockDim.x * blockIdx.x + threadIdx.x;
  int j = ri / c.nRays_line;
  int i = ri - j * c.nRays_line;
  float ray_x1 = i * c.dx_rays - c.R_rays;
  float ray_x2 = j * c.dx_rays - c.R_rays;

  if (dst(ray_x1, ray_x2) <= c.R_rays) {
    float sum_x1 = 0.0;
    float sum_x2 = 0.0;
    for (int i = 0; i < c.nMicrolenses; i++) {
      float m_x1 = ray_x1 - uls[i].x1 - (uls[i].v1 * t);
      float m_x2 = ray_x2 - uls[i].x2 - (uls[i].v2 * t);
      float ri = uls[i].m * dst2_inv(m_x1, m_x2);
      sum_x1 += m_x1 * ri;
      sum_x2 += m_x2 * ri;
    }
    ray_x1 = (1 - c.gamma) * ray_x1 - c.sigma_c * ray_x1 - sum_x1;
    ray_x2 = (1 + c.gamma) * ray_x2 - c.sigma_c * ray_x2 - sum_x2;

    if (c.output_rays) {
        rays[ri].x1 = ray_x1;
        rays[ri].x2 = ray_x2;    
    }

    int w = lrintf((ray_x1 - c.image_y1_left) / c.image_pixel_y1_size);
    int h = lrintf((ray_x2 - c.image_y2_bottom) / c.image_pixel_y2_size);
    if (w >= 0 && w < c.image_width && h >= 0 && h < c.image_height) atomicAdd(&image[w * c.image_height + h], 1);


    for (int i = 0; i < c.nLCsteps; i++) {
        float lc_y1 = lc[i + 0 * c.nLCsteps];
        float lc_y2 = lc[i + 1 * c.nLCsteps];
        float d = dst(lc_y1 - ray_x1, lc_y2 - ray_x2);
        float sigma = 0.1;
        float sigma2 = 0.01;
        if (d < 4 * sigma) {
            float val = expf(- d * d / sigma2);
            //atomicAdd(&lc[i + 2 * c.nLCsteps], 1.0); // Normalization
            //atomicAdd(&lc[i + 3 * c.nLCsteps], val); // Amplitude value, non-normalized
        }
    }
  }
}

__global__ void calculateLCs(const Configuration c, int *image, float *lc) {
  int w = blockIdx.x * blockDim.x + threadIdx.x;
  int h = blockIdx.y * blockDim.y + threadIdx.y;


  float r_x1 = w * c.image_pixel_y1_size + c.image_y1_left;
  float r_x2 = h * c.image_pixel_y2_size + c.image_y2_bottom;

  for (int i = 0; i < c.nLCsteps; i++) {
    float lc_y1 = lc[i + 0 * c.nLCsteps];
    float lc_y2 = lc[i + 1 * c.nLCsteps];
    float d = dst(lc_y1 - r_x1, lc_y2 - r_x2);
    float sigma = 0.1;
    float sigma2 = 0.01;
    if (d < 4 * sigma) {
        float v = expf(- d * d / sigma2);
        float val = image[w * c.image_height + h] * v;
        atomicAdd(&lc[i + 2 * c.nLCsteps], v); // Normalization
        atomicAdd(&lc[i + 3 * c.nLCsteps], val); // Amplitude value, non-normalized
    }
    //factorex = exp(-dist2 / sigsq2);
    //dum = pix[ix + i1 + (iy + i2) * pix_dim1];
    //value += dum * factorex;
    //normfac += factorex;
  }

}